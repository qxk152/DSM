#include "hip/hip_runtime.h"
#include "defs.h"
#include "gpu_match.cuh"
#include "cuda_helpers.h"
#include "order.h"
#include "join.cuh"
#include "memManag.cuh"
#include "res_table.hpp"

#include <hip/hip_cooperative_groups.h>

#include <cstdio>
#include <ctime>
#include <chrono>

__global__ void
warmup()
{
  int a = IDX * 1;
}

__device__ void
intersect(Arg_t *__restrict__ arg)
{
  __shared__ offtype off[WARP_PER_BLOCK];
  __shared__ offtype off_ed[WARP_PER_BLOCK];
  __shared__ bool found[WARP_PER_BLOCK][WARP_SIZE];
  __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

  __shared__ int cnt[WARP_PER_BLOCK];

  if (LID == 0)
  {
    cnt[WID] = 0;
    off[WID] = 0;
    off_ed[WID] = arg->set1_size;
  }
  __syncwarp();
  // vtype v;
  // bool found = false;

  while (off[WID] < off_ed[WID])
  {
    // off[WID] + lid = off[WID] + LID;
    v[WID][LID] = UINT32_MAX;
    if (off[WID] + LID < off_ed[WID])
      v[WID][LID] = arg->set1[off[WID] + LID];
    // else
    // v[WID][LID] = UINT32_MAX;
    __syncwarp();
    found[WID][LID] = false;
    if (v[WID][LID] != UINT32_MAX)
    {
      int res = lower_bound(arg->set2, arg->set2_size, v[WID][LID]);
      if (res != UINT32_MAX && arg->set2[res] == v[WID][LID])
        found[WID][LID] = true;
      // #pragma unroll 8
      //       for (int i = 0; i < arg->set2_size; ++i)
      //       {
      //         // found[WID][LID] |= (arg->set2[i] == v[WID][LID]);
      //         if (arg->set2[i] == v[WID][LID])
      //         {
      //           found[WID][LID] = true;
      //           break;
      //         }
      //         else if (arg->set2[i] > v[WID][LID])
      //         {
      //           break;
      //         }
      //       }
    }
    __syncwarp();
    if (found[WID][LID])
    {
      int mask = __activemask();
      int size = __popc(mask);
      int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
      int pos = cnt[WID];
      // if (pos + rank > C_NUM_CAN_UB)
      // printf("pos + rank = %d\n", pos + rank);
      arg->res[pos + rank] = v[WID][LID];

      if (rank == 0)
        cnt[WID] += size;
    }
    __syncwarp();
    if (LID == 0)
      off[WID] += warpSize;
    __syncwarp();
  }
  __syncwarp();
  if (LID == 0)
    arg->res_size[0] = cnt[WID];
  __syncwarp();
}

__device__ void
get_new_v(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ stk, int *__restrict__ cur,
    StealingArgs *__restrict__ stealing_args, long long &start_clk,
    OrderGPU *__restrict__ order_obj,

    int start_level,

    uint32_t *__restrict__ compact_encodings_,
    int num_blocks,

    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows,

    bool *__restrict__ ret, vtype *__restrict__ cur_res, int *order_id)
{
  if (stk->level == start_level)
  {
    __shared__ int next_pos[WARP_PER_BLOCK];
    // __shared__ vtype u[WARP_PER_BLOCK][WARP_SIZE];
    // int next_pos;
    if (LID == 0)
      next_pos[WID] = atomicAdd(cur, 1);
    __syncwarp();
    // next_pos = __shfl_sync(FULL_MASK, next_pos, 0);

    if (next_pos[WID] >= num_initial_task_table_rows)
    {
      if (LID == 0)
        *ret = false;
      __syncwarp();
      return;
    }

    {
      if (LID == 0)
      {
        *order_id = initial_task_table_[next_pos[WID] * (start_level + 1) + 0];
      }
      // int l = LID; // one lane -- one level
      else if (LID <= start_level + 1)
      { //对于其他 lane（1 到 start_level+1），每个线程读取对应位置的候选顶点，并将结果存入局部当前结果数组 cur_res，同时将调用栈中对应层的迭代器初始化为 0，并设置候选数为 1。
        // u[WID][LID] = order_obj->v_orders_[order_id * C_NUM_VQ + l];
        // u[WID][LID] = v_order[l];
        stk->iter[LID] = 0;
        cur_res[LID - 1] = initial_task_table_[next_pos[WID] * (start_level + 1) + LID];
        // cur_res[u[WID][LID]] = initial_task_table_[next_pos[WID] * C_NUM_VQ + u[WID][LID]];
        // stk->candidates_[l * C_NUM_CAN_UB + 0] = initial_task_table_[next_pos * C_NUM_VQ + u];
        stk->num_candidates_[LID] = 1;
      }
    }
    __syncwarp();
  }
  else // level < C_NUM_VQ - 1
  {
    __shared__ vtype u[WARP_PER_BLOCK];
    __shared__ int cnt[WARP_PER_BLOCK];
    __shared__ Arg_t arg[WARP_PER_BLOCK];
    __shared__ vtype mapped_vs_[WARP_PER_BLOCK][WARP_SIZE];

    __shared__ int min_i[WARP_PER_BLOCK];
    __shared__ int min_nbrs[WARP_PER_BLOCK];
    __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

    if (LID < C_NUM_VQ)
      flag[WID][LID] = false;
    __syncwarp();

    if (LID == 0)
      u[WID] = order_obj->v_orders_[(*order_id) * C_NUM_VQ + stk->level];
    // u[WID] = v_order[stk->level];
    __syncwarp();
    // vtype u = v_order[stk->level];

    // if (LID == 0)
    //   *cnt = 0;
    // __syncwarp();

    // if (LID == 0)
    // printf("num_bn: %d\n", order_obj->num_backward_neighbors_[u]);
    // __syncwarp();

    if (LID < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]])
    {
      vtype u_back = order_obj->backward_neighbors_[(*order_id) * C_NUM_VQ * C_NUM_VQ + u[WID] * C_NUM_VQ + LID];
      // int l_back = order_obj->u2l_[u_back];
      // vtype v_back = cur_res[u_back];
      // vtype v_back = stk->candidates_[l_back * C_NUM_CAN_UB + stk->iter[l_back]];
      // mapped_vs_[WID][LID] = cur_res[u_back];
      mapped_vs_[WID][LID] = cur_res[order_obj->u2ls_[(*order_id) * C_NUM_VQ + u_back]];
    }
    __syncwarp();

    if (LID == 0)
    {
      if (order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]] == 1)
      {
        min_i[WID] = 0;
      }
      else
      {
        min_i[WID] = 0;
        min_nbrs[WID] = d_degs_[mapped_vs_[WID][0]];
        for (int i = 1; i < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++i)
        {
          if (d_degs_[mapped_vs_[WID][i]] < min_nbrs[WID])
          {
            min_i[WID] = i;
            min_nbrs[WID] = d_degs_[mapped_vs_[WID][i]];
          }
        }
      }
    }
    __syncwarp();

    if (LID == 0)
    {
      // printf("mapped_vs_[0] = %d\n", mapped_vs_[0]);
      arg[WID].res = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
      arg[WID].res_size = stk->num_candidates_ + stk->level;
      arg[WID].res_size[0] = d_degs_[mapped_vs_[WID][min_i[WID]]];
      flag[WID][min_i[WID]] = true;
    }
    __syncwarp();
    for (int i = 1; i < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++i)
    {
      if (LID == 0)
      {
        min_nbrs[WID] = UINT32_MAX;
        for (int j = 0; j < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++j)
        {
          if (flag[WID][j])
            continue;
          if (d_degs_[mapped_vs_[WID][j]] < min_nbrs[WID])
          {
            min_i[WID] = j;
            min_nbrs[WID] = d_degs_[mapped_vs_[WID][j]];
          }
        }

        arg[WID].set1 = arg[WID].res;
        arg[WID].set1_size = arg[WID].res_size[0];
        arg[WID].set2 = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
        arg[WID].set2_size = d_degs_[mapped_vs_[WID][min_i[WID]]];
        // arg[WID].res = can + (stk->level - start_level - 1) * C_NUM_CAN_UB;
        arg[WID].res = stk->candidates_ + stk->level * C_NUM_CAN_UB;
        arg[WID].res_size = stk->num_candidates_ + stk->level;
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      intersect(&arg[WID]);
    }

    if (LID == 0)
      cnt[WID] = 0;
    __syncwarp();

    __shared__ offtype off[WARP_PER_BLOCK];
    __shared__ offtype off_ed[WARP_PER_BLOCK];
    __shared__ bool dup[WARP_PER_BLOCK][WARP_SIZE];
    __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

    if (LID == 0)
    {
      off[WID] = 0;
      off_ed[WID] = arg[WID].res_size[0];
    }
    __syncwarp();
    // vtype v;
    // bool dup = false;
    while (off[WID] < off_ed[WID])
    {
      int my_off = off[WID] + LID;
      if (my_off < off_ed[WID])
        v[WID][LID] = arg[WID].res[my_off];
      else
        v[WID][LID] = UINT32_MAX;
      __syncwarp();
      dup[WID][LID] = false;
      if (v[WID][LID] != UINT32_MAX)
        for (int l = 0; l < stk->level; ++l)
        {
          if (v[WID][LID] == cur_res[l])
          // if (v[WID][LID] == cur_res[v_order[l]])
          {
            dup[WID][LID] = true;
            break;
          }
          // if (v[WID][LID] == stk->candidates_[l * C_NUM_CAN_UB + stk->iter[l]])
          // {
          //   dup = true;
          //   break;
          // }
        }
      __syncwarp();
      if (v[WID][LID] != UINT32_MAX &&
          !dup[WID][LID] &&
          (compact_encodings_[u[WID] * C_COL_LEN + v[WID][LID] / BLK_SIZE] & (1 << (v[WID][LID] % BLK_SIZE))))
      {
        int mask = __activemask();
        int size = __popc(mask);
        int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
        int my_pos = cnt[WID] + rank;
        // can[(stk->level - start_level - 1) * C_NUM_CAN_UB + my_pos] = v[WID][LID];
        stk->candidates_[stk->level * C_NUM_CAN_UB + my_pos] = v[WID][LID];
        if (rank == 0)
          cnt[WID] += size;
      }
      __syncwarp();
      if (LID == 0)
        off[WID] += warpSize;
      __syncwarp();
    }
    __syncwarp();
    if (LID == 0)
    {
      if (cnt[WID] == 0)
        *ret = false;
      else
      {
        stk->iter[stk->level] = 0;
        stk->num_candidates_[stk->level] = cnt[WID];
        // cur_res[u[WID]] = can[(stk->level - start_level - 1) * C_NUM_CAN_UB + 0];
        // cur_res[u[WID]] = stk->candidates_[stk->level * C_NUM_CAN_UB + 0];
        cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + 0];
      }
    }
    __syncwarp();
  }
}

__device__ void
steal(
    CallStack *__restrict__ stk, StealingArgs *__restrict__ stealing_args, bool *__restrict__ __restrict__ ret, long long &start_clk, vtype *v_order,
    int *queue_arr_,
    vtype *__restrict__ cur_res, int *order_id)
{
  __shared__ bool flag[WARP_PER_BLOCK];

  if (LID == 0)
  {
    stk->stealed_task = false;
    flag[WID] = stealing_args->queue->dequeue(queue_arr_, C_STOP_LEVEL);
  }
  __syncwarp();

  if (flag[WID])
  {
    if (LID < C_STOP_LEVEL && LID > 0)
    {
      if (queue_arr_[LID] != DeletionMarker<int>::val - 1)
      {
        stk->iter[LID] = 0;
        stk->num_candidates_[LID] = 1;
        // stk->candidates_[LID * C_NUM_CAN_UB + 0] = queue_arr_[LID];
        // cur_res[v_order[LID]] = queue_arr_[LID];
        cur_res[LID] = queue_arr_[LID];
        atomicMax(&stk->level, LID);
      }
    }
    else if (LID == 0)
    {
      *order_id = queue_arr_[0];
    }
    __syncwarp();
    if (LID == 0)
    {
      stk->iter[stk->level + 1] = 0;
      stk->num_candidates_[stk->level + 1] = 0;
      stk->stealed_task = true;
    }
    __syncwarp();
  }
  else
  {
    if (LID == 0)
      *ret = false;
    __syncwarp();
  }
  if (LID == 0)
    start_clk = clock64();
  __syncwarp();
  // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);

  // if (LID == 0)
  // {
  //   stk->stealed_task = false;
  //   bool flag = stealing_args->queue->dequeue(queue_arr_, C_STOP_LEVEL);
  //   if (flag)
  //   {
  //     for (int i = 0; i < C_STOP_LEVEL; ++i)
  //     {
  //       int val = queue_arr_[i];
  //       if (val != DeletionMarker<int>::val - 1)
  //       {
  //         stk->iter[i] = 0;
  //         stk->num_candidates_[i] = 1;
  //         // stk->candidates_[i * C_NUM_CAN_UB + 0] = val;
  //         cur_res[v_order[i]] = val;
  //         stk->level = i;
  //       }
  //       else
  //       {
  //         stk->iter[i] = 0;
  //         stk->num_candidates_[i] = 0;
  //         break;
  //       }
  //     }
  //     stk->stealed_task = true;
  //   }
  //   else
  //   {
  //     *ret = false;
  //   }
  // }
  // __syncwarp();
  // if (LID == 0)
  //   start_clk = clock64();
  // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
}

__device__ void
match(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ stk, int *__restrict__ cur, unsigned long long *__restrict__ count,
    StealingArgs *__restrict__ stealing_args, long long &start_clk,
    OrderGPU *__restrict__ order_obj, int start_level, int *__restrict__ queue_arr_,
    uint32_t *__restrict__ compact_encodings_, int num_blocks,
    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows,

    bool *ret, vtype *cur_res)
{ // initial_task_table_ = d_u_candidate_vs_
  // uint32_t &level = stk->level;
  // can = candidates[wid]

  __shared__ int s_order_id[WARP_PER_BLOCK];

  while (true)
  { //首先检查调用栈当前层是否处于起始层
    if (stk->level == start_level)
    {
      if (LID == 0)
        *ret = true;
      __syncwarp();

      // steal(stk, stealing_args, ret, start_clk, v_order, queue_arr_, cur_res);
      steal(stk, stealing_args, ret, start_clk, order_obj->v_orders_, queue_arr_, cur_res, &s_order_id[WID]);
      if (*ret == false) // nothing to steal
      {
        if (LID == 0)
          *ret = true;
        __syncwarp();
        get_new_v(
            d_degs_, d_offsets_, d_nbrs_,
            stk, cur, stealing_args, start_clk,
            order_obj, start_level,
            compact_encodings_, num_blocks,
            initial_task_table_, num_initial_task_table_rows,
            ret, cur_res, &s_order_id[WID]);
        __syncwarp();
      }
      else
      {
        if (LID == 0)
        {
          *ret = true;
        }
        __syncwarp();
      }
      if (*ret == false) // no more candidates
      {
        break;
      }
      else
      {
        if (LID == 0)
        {
          stk->level++;
          stk->iter[stk->level] = 0;
          stk->num_candidates_[stk->level] = 0;
        }
        __syncwarp();
      }
    }
    else if (stk->level == C_NUM_VQ - 1) // final, intersect and return
    {
      __shared__ Arg_t arg[WARP_PER_BLOCK];
      __shared__ vtype mapped_vs_[WARP_PER_BLOCK][WARP_SIZE];
      __shared__ int cnt[WARP_PER_BLOCK];

      __shared__ int min_i[WARP_PER_BLOCK];
      __shared__ int min_nbrs[WARP_PER_BLOCK];
      __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

      if (LID < C_NUM_VQ)
        flag[WID][LID] = false;
      __syncwarp();

      vtype u = order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + stk->level];
      // vtype u = v_order[stk->level];

      if (LID < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u])
      {
        vtype u_back = order_obj->backward_neighbors_[s_order_id[WID] * C_NUM_VQ * C_NUM_VQ + u * C_NUM_VQ + LID];
        // vtype u_back = order_obj->backward_neighbors_[u * C_NUM_VQ + LID];
        // int l_back = order_obj->u2l_[u_back];
        // vtype v_back = cur_res[u_back];
        // vtype v_back = stk->candidates_[l_back * C_NUM_CAN_UB + stk->iter[l_back]];
        // mapped_vs_[WID][LID] = cur_res[u_back];
        mapped_vs_[WID][LID] = cur_res[order_obj->u2ls_[s_order_id[WID] * C_NUM_VQ + u_back]];
      }
      __syncwarp();

      if (LID == 0)
      {
        if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u] == 1)
        {
          min_i[WID] = 0;
        }
        else
        {
          min_i[WID] = 0;
          min_nbrs[WID] = d_degs_[mapped_vs_[WID][0]];
          for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++i)
          {
            if (d_degs_[mapped_vs_[WID][i]] < min_nbrs[WID])
            {
              min_i[WID] = i;
              min_nbrs[WID] = d_degs_[mapped_vs_[WID][i]];
            }
          }
        }
      }
      __syncwarp();

      if (LID == 0)
      {
        arg[WID].res = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
        arg[WID].res_size = stk->num_candidates_ + stk->level;
        arg[WID].res_size[0] = d_degs_[mapped_vs_[WID][min_i[WID]]];
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++i)
      {
        if (LID == 0)
        {
          min_nbrs[WID] = UINT32_MAX;
          for (int j = 0; j < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++j)
          {
            if (flag[WID][j])
              continue;
            if (d_degs_[mapped_vs_[WID][j]] < min_nbrs[WID])
            {
              min_i[WID] = j;
              min_nbrs[WID] = d_degs_[mapped_vs_[WID][j]];
            }
          }

          arg[WID].set1 = arg[WID].res;
          arg[WID].set1_size = arg[WID].res_size[0];
          arg[WID].set2 = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
          arg[WID].set2_size = d_degs_[mapped_vs_[WID][min_i[WID]]];
          // arg[WID].res = can_wid + (stk->level - start_level - 1) * C_NUM_CAN_UB;
          arg[WID].res = stk->candidates_ + stk->level * C_NUM_CAN_UB;
          arg[WID].res_size = stk->num_candidates_ + stk->level;
          flag[WID][min_i[WID]] = true;
        }
        __syncwarp();
        intersect(&arg[WID]);
      }

      if (LID == 0)
        cnt[WID] = 0;
      __syncwarp();
      // int pos = d_enc_pos_u_[u];

      __shared__ offtype off_ed[WARP_PER_BLOCK];
      __shared__ offtype off[WARP_PER_BLOCK];
      __shared__ bool dup[WARP_PER_BLOCK][WARP_SIZE];
      __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

      off[WID] = 0;
      off_ed[WID] = arg[WID].res_size[0];
      // vtype v;
      dup[WID][LID] = false;
      while (off[WID] < off_ed[WID])
      {
        if (off[WID] + LID < off_ed[WID])
          v[WID][LID] = arg[WID].res[off[WID] + LID];
        else
          v[WID][LID] = UINT32_MAX;
        __syncwarp();
        dup[WID][LID] = false;
        if (v[WID][LID] != UINT32_MAX)
          for (int l = 0; l < stk->level; ++l)
          {
            if (v[WID][LID] == cur_res[l])
            {
              dup[WID][LID] = true;
              break;
            }

            // dup[WID][LID] |= (v[WID][LID] == cur_res[v_order[l]]);
            // if (dup[WID][LID])
            //   break;
            // if (v[WID][LID] == cur_res[v_order[l]])
            // dup[WID][LID] = true;
            // if (v == stk->candidates_[l * C_NUM_CAN_UB + stk->iter[l]])
            // dup[WID][LID] = true;
          }
        __syncwarp();
        if (v[WID][LID] != UINT32_MAX &&
            !dup[WID][LID] &&
            (compact_encodings_[u * C_COL_LEN + v[WID][LID] / BLK_SIZE] & (1 << (v[WID][LID] % BLK_SIZE))))
        // if (v != UINT32_MAX && !dup[WID][LID] && (compact_encodings_[v * num_blocks + pos / BLK_SIZE] & (1 << (pos % BLK_SIZE))))
        {
          int mask = __activemask();
          int size = __popc(mask);
          int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
          int my_pos = cnt[WID] + rank;
          // can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + my_pos] = v[WID][LID];
          stk->candidates_[stk->level * C_NUM_CAN_UB + my_pos] = v[WID][LID];
          if (rank == 0)
            cnt[WID] += size;
        }
        __syncwarp();
        if (LID == 0)
          off[WID] += warpSize;
        __syncwarp();
      }
      if (LID == 0)
      {
        *count += cnt[WID];
        stk->level--;
        stk->iter[stk->level]++;
        // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
        // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
      }
      __syncwarp();
    }
    else // intermediate levels
    {
      __shared__ int is_timeout[WARP_PER_BLOCK];
      // int is_timeout;
      if (LID == 0)
        is_timeout[WID] = stk->level == C_STOP_LEVEL - 1 && ELAPSED_TIME(start_clk) > TIMEOUT && !stk->stealed_task;
      __syncwarp();
      // is_timeout = __shfl_sync(FULL_MASK, is_timeout, 0);

      if (stk->num_candidates_[stk->level] == 0) // top-down, get new candidates and keep going down. Or return.
      {
        if (LID == 0)
          *ret = true;
        __syncwarp();
        get_new_v(
            d_degs_, d_offsets_, d_nbrs_,
            stk, cur, stealing_args, start_clk,
            order_obj, start_level,
            compact_encodings_, num_blocks,
            initial_task_table_, num_initial_task_table_rows,
            ret, cur_res, &s_order_id[WID]);
        if (*ret == false) // no candidates, return
        {
          if (LID == 0)
          {
            stk->iter[stk->level] = 0;
            stk->level--;
            stk->iter[stk->level]++;
            // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
            // cur_res[order_obj->v_orders_[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
            cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          }
          __syncwarp();
        }
        else // candidates found, go down.
        {
          if (LID == 0)
          {
            stk->iter[stk->level] = 0;
            stk->level++;
            stk->iter[stk->level] = 0;
            stk->num_candidates_[stk->level] = 0;
          }
          __syncwarp();
        }
      }
      else if (stk->iter[stk->level] == stk->num_candidates_[stk->level]) // end of this level, return.
      {
        if (LID == 0)
        {
          stk->iter[stk->level] = 0;
          stk->num_candidates_[stk->level] = 0;

          stk->level--;
          stk->iter[stk->level]++;
          // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
          // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        }
        __syncwarp();
        if (stk->level == start_level)
        {
          if (LID == 0)
            start_clk = clock64();
          __syncwarp();
          // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
        }
      }
      else // not end, map next v, then keep going down.
      {
        if (!is_timeout[WID])
        {
          if (LID == 0)
          {
            stk->level++;
            stk->iter[stk->level] = 0;
            stk->num_candidates_[stk->level] = 0;
          }
          __syncwarp();
        }
        else // timeout, split task into queue.
        {
          __shared__ bool enqueue_succ[WARP_PER_BLOCK];
          // int enqueue_succ = false;
          if (LID == 0)
            queue_arr_[0] = s_order_id[WID];
          else if (LID < C_STOP_LEVEL)
          {
            // queue_arr_[LID] = cur_res[v_order[LID]];
            queue_arr_[LID] = cur_res[LID - 1];
          }
          __syncwarp();
          if (LID == 0)
          {
            enqueue_succ[WID] = false;
            // for (int i = 0; i < C_STOP_LEVEL - 1; ++i)
            // {
            //   queue_arr_[i] = cur_res[v_order[i]];
            //   // queue_arr_[i] = stk->candidates_[i * C_NUM_CAN_UB + 0];

            //   // queue_arr_[i] = cur_res[v_order[i]];
            //   // if (stk->num_candidates_[i])
            //   //   // queue_arr_[i] = cur_res[v_order[i]];
            //   //   queue_arr_[i] = stk->candidates_[i * C_NUM_CAN_UB + stk->iter[i]];
            //   // else
            //   //   queue_arr_[i] = DeletionMarker<int>::val - 1;
            // }
            queue_arr_[C_STOP_LEVEL] = DeletionMarker<int>::val - 1;
            // queue_arr_[C_STOP_LEVEL - 1] = DeletionMarker<int>::val - 1;
#pragma unroll 2
            for (; stk->iter[stk->level] < stk->num_candidates_[stk->level]; ++stk->iter[stk->level])
            {
              // queue_arr_[C_STOP_LEVEL - 1] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
              // queue_arr_[C_STOP_LEVEL - 1] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              queue_arr_[C_STOP_LEVEL] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              enqueue_succ[WID] = stealing_args->queue->enqueue(queue_arr_, C_STOP_LEVEL + 1);
              if (!enqueue_succ[WID])
                break;
            }
          }
          __syncwarp();
          // enqueue_succ = __shfl_sync(FULL_MASK, enqueue_succ, 0);
          if (enqueue_succ[WID])
          {
            stk->num_candidates_[stk->level] = 0;
            stk->iter[stk->level] = 0;
            if (stk->level > start_level)
            {
              if (LID == 0)
              {
                stk->level--;
                stk->iter[stk->level]++;
                // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
                cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
                // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
                // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              }
              __syncwarp();
            }
          }
          else
          {
            if (LID == 0)
              start_clk = clock64();
            __syncwarp();
            // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
          }
        }
      }
    }
  }
  __syncwarp();
}

__global__ void
parallel_match_kernel(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ call_stack, int *__restrict__ cur, unsigned long long *__restrict__ res,
    Queue *__restrict__ queue, OrderGPU *__restrict__ order_obj, int num_orders,

    int start_level,

    uint32_t *__restrict__ compact_encodings_,
    int num_blocks,
    vtype *__restrict__ d_u_candidate_vs_, numtype *__restrict__ d_num_u_candidate_vs_,

    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows)
{ // initial_task_table_ = d_u_candidate_vs_
  queue->init();

  // __shared__ OrderGPU s_order_obj(num_orders);
  __shared__ CallStack stk[WARP_PER_BLOCK];

  __shared__ unsigned long long s_count[WARP_PER_BLOCK]; // total count of results this warp
  __shared__ vtype cur_res[WARP_PER_BLOCK][MAX_VQ];      // current result, this warp
  // __shared__ vtype s_v_order[MAX_VQ];                    // shared across this block

  // for match()
  __shared__ bool ret[WARP_PER_BLOCK]; // return value of get_new_v(), this warp

  // for steal.queue
  __shared__ int queue_arr_[WARP_PER_BLOCK][MAX_VQ];
  __shared__ StealingArgs s_stealing_args;
  __shared__ long long start_clk[WARP_PER_BLOCK];

  s_stealing_args.queue = queue;

  // if (TID == 0)
  // {
  // s_order_obj = *order_obj;
  // }

  // if (TID < C_NUM_VQ)
  // {
  // s_v_order[TID] = s_order_obj.v_order_[TID];
  // }
  __syncthreads();

  if (LID == 0)
  {
    stk[WID].candidates_ = call_stack[WID_G].candidates_;
    // stk[WID] = call_stack[WID_G];
    stk[WID].level = start_level;
    stk[WID].num_candidates_[start_level] = 0;
    s_count[WID] = 0;
  }
  __syncwarp();

  // long long st = clock64();
  if (LID == 0)
    start_clk[WID] = clock64();
  __syncwarp();
  match(
      d_degs_, d_offsets_, d_nbrs_,
      &stk[WID], cur,
      &s_count[WID], &s_stealing_args,
      start_clk[WID], order_obj,

      start_level, queue_arr_[WID],

      compact_encodings_,
      num_blocks,

      initial_task_table_, num_initial_task_table_rows,

      &ret[WID], cur_res[WID]);
  __syncwarp();
  // long long ed = clock64();

  if (LID == 31)
  {
    res[WID_G] = s_count[WID];
    // printf("bid: %d, WID: %d, time: %lf(ms)\n", BID, WID_G, (ed - start_clk) / (1.0 * CLOCK_RATE));
    // printf("WID: %d, count: %lu\n", WID_G, s_count[WID]);
  }
  __syncwarp();
}
//d_res_table_old 存储{0, 5, 6, 1, 5, 6, 2, 5, 6, 3, 5, 6} num_res_old存储 结果为4
__global__ void
vertexJoinBFS(
    offtype *d_offsets_, vtype *d_neighbors_, degtype *d_degree_,
    OrderGPU *order_obj,  
    int level, // size of one row = level + 1
    vtype *intersect_temp_storage, numtype *num_intersect_temp_storage,

    uint32_t *d_encodings_, numtype num_blocks, int *d_enc_pos_u_,
    vtype *d_res_table_old, numtype num_res_old,
    vtype *d_res_table, numtype *num_res_new,
    int *exceed)
{
  // int tid = threadIdx.x;
  // int bid = blockIdx.x;
  // int idx = tid + bid * blockDim.x;
  // int wid = tid >> 5;
  // int lid = tid & 31;
  // int wid_g = idx >> 5;

  __shared__ int s_row[WARP_PER_BLOCK];
  __shared__ vtype s_cur_res[WARP_PER_BLOCK][MAX_VQ];
  __shared__ vtype mapped_vs_[WARP_PER_BLOCK][MAX_VQ];
  __shared__ Arg_t arg[WARP_PER_BLOCK];
  __shared__ int warp_pos[WARP_PER_BLOCK];
  // __shared__ int s_v_order_[MAX_VQ];
  // __shared__ int s_num_bn[MAX_VQ];
  // __shared__ vtype s_bn[MAX_VQ][MAX_VQ];

  __shared__ int s_order_id[WARP_PER_BLOCK];
  __shared__ vtype s_u[WARP_PER_BLOCK];
  __shared__ int num_warps;
  __shared__ int block_iter_cnt[WARP_PER_BLOCK];
  // __shared__ int s_pos_u[MAX_VQ];
  // __shared__ int s_orders_[32][MAX_VQ];

  __shared__ int min_i[WARP_PER_BLOCK];
  __shared__ int min_nbrs[WARP_PER_BLOCK];
  __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

  // if (LID < C_NUM_VQ)
  //   flag[WID][LID] = false;
  // __syncwarp();

  // int block_iter_cnt = 0;

  if (LID == 0) //LID 是线程在warp中的id WID是当前的warp的id  初始化当前warp在第几层
    block_iter_cnt[WID] = 0;
  __syncwarp();

  if (TID == 0)
    num_warps = blockDim.x / warpSize * gridDim.x;
  __syncthreads();
  
  
  if (TID < C_NUM_VQ)
  {
    // s_pos_u[TID] = d_enc_pos_u_[TID];
    // s_num_bn[TID] = order_obj->num_backward_neighbors_[TID];
    // s_v_order_[TID] = order_obj->v_orders_[TID];
  }
  __syncthreads();
  // if (TID < C_NUM_VQ * C_NUM_VQ)
  //   s_bn[TID / C_NUM_VQ][TID % C_NUM_VQ] = order_obj->backward_neighbors_[TID];
  // __syncthreads();

  // vtype u = s_v_order_[level];
  // 所有 warp 以一个步长为 num_warps 的方式分摊所有匹配结果
  //block_iter_cnt 记录warp迭代了几次
  
  while (WID_G + block_iter_cnt[WID] * num_warps < num_res_old)
  {
    if (LID < C_NUM_VQ)
      flag[WID][LID] = false;
    __syncwarp();
    if (LID == 0)
    {
      s_row[WID] = WID_G + block_iter_cnt[WID] * num_warps; // 当前warp处理哪个orderID
      s_order_id[WID] = d_res_table_old[s_row[WID] * (level + 1)];
      s_u[WID] = order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + level]; // 这是要匹配的查询顶点
    }
    __syncwarp();
    if (LID < level)
      s_cur_res[WID][LID] = d_res_table_old[s_row[WID] * (level + 1) + 1 + LID];
    __syncwarp();

    // if (LID < s_num_bn[u])
    if (LID < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]])
    { //如果查询顶点有 2 个后向邻居，那么只有 LID 为 0 和 1 的线程参与处理。 数组中取出当前查询顶点对应的第 LID 个后向邻居。
      vtype u_back = order_obj->backward_neighbors_[s_order_id[WID] * C_NUM_VQ * C_NUM_VQ + s_u[WID] * C_NUM_VQ + LID];
      mapped_vs_[WID][LID] = s_cur_res[WID][order_obj->u2ls_[s_order_id[WID] * C_NUM_VQ + u_back]];
      // 假设 s_cur_res[WID] 存储的是 {5, 6}，那么如果LID= 0，就表示后向邻居 u_back 对应的是更新边的起始顶点 5；如果返回 1，则对应更新边的顶点 6。
      // mapped_vs_[WID][LID] = v_back;
    }
    __syncwarp();
    
    if (LID == 0)
    {
      min_i[WID] = 0;
      // if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + level] == 1)
      // {
      // min_i[WID] = 0;
      // }
      if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]] > 1)
      {
        // min_i[WID] = 0;
        min_nbrs[WID] = d_degree_[mapped_vs_[WID][0]];
        for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++i)
        {
          if (d_degree_[mapped_vs_[WID][i]] < min_nbrs[WID])
          {
            min_i[WID] = i;
            min_nbrs[WID] = d_degree_[mapped_vs_[WID][i]]; // 数据图顶点的度
          }
        }
      }
    } // 每一个warp中的第一个线程选择一个度最小的顶点
    __syncwarp();
   
    if (LID == 0)
    {
      // printf("mapped_vs_[0] = %d\n", mapped_vs_[0]); 就是表示当前 warp 中选择的那个后向邻居对应的顶点值
      arg[WID].res = d_neighbors_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];  // 存储的是映射的最小度顶点的令居集合
      arg[WID].res_size = num_intersect_temp_storage + WID_G;
      arg[WID].res_size[0] = d_degree_[mapped_vs_[WID][min_i[WID]]];
      flag[WID][min_i[WID]] = true;
    }
    __syncwarp();
    for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++i)
    {
      if (LID == 0)
      {
        min_nbrs[WID] = UINT32_MAX;
        for (int j = 0; j < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++j)
        {
          if (flag[WID][j])
            continue;
          if (d_degree_[mapped_vs_[WID][j]] < min_nbrs[WID])
          {
            min_i[WID] = j;
            min_nbrs[WID] = d_degree_[mapped_vs_[WID][j]];
          }
        }

        arg[WID].set1 = arg[WID].res;
        arg[WID].set1_size = arg[WID].res_size[0];
        arg[WID].set2 = d_neighbors_ + d_offsets_[mapped_vs_[WID][min_i[WID]]]; // 是两个最小度后向邻居的邻居候选集取交集
        arg[WID].set2_size = d_degree_[mapped_vs_[WID][min_i[WID]]];
        // arg[WID].res = can + (stk->level - start_level - 1) * C_NUM_CAN_UB;
        arg[WID].res = intersect_temp_storage + WID_G * C_NUM_CAN_UB;
        arg[WID].res_size = num_intersect_temp_storage + WID_G;
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      intersect(&arg[WID]);
    }
    // if (LID == 0)
    // {
    //   arg[WID].res = d_neighbors_ + d_offsets_[mapped_vs_[WID][0]];
    //   arg[WID].res_size = num_intersect_temp_storage + WID_G;
    //   arg[WID].res_size[0] = d_degree_[mapped_vs_[WID][0]];
    // }
    // __syncwarp();
    // for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + level]; ++i)
    // // for (int i = 1; i < s_num_bn[u]; ++i)
    // {
    //   if (LID == 0)
    //   {
    //     arg[WID].set1 = arg[WID].res;
    //     arg[WID].set1_size = arg[WID].res_size[0];
    //     arg[WID].set2 = d_neighbors_ + d_offsets_[mapped_vs_[WID][i]];
    //     arg[WID].set2_size = d_degree_[mapped_vs_[WID][i]];
    //     arg[WID].res = intersect_temp_storage + WID_G * C_NUM_CAN_UB;
    //     arg[WID].res_size = num_intersect_temp_storage + WID_G;
    //   }
    //   __syncwarp();
    //   intersect(&arg[WID]);
    // }
    // int pos = s_pos_u[s_u[WID]];

    offtype off = 0;
    offtype off_ed = arg[WID].res_size[0];
    vtype v;
    bool dup = false;
    
    while (off < off_ed)
    {
      int my_off = off + LID;
      if (my_off < off_ed)
        v = arg[WID].res[my_off];
      else
        v = UINT32_MAX;
      __syncwarp();
      dup = false;
      if (v != UINT32_MAX)
        for (int l = 0; l < level; ++l)
        {
          // if (v == s_cur_res[WID][order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + l]])
          // if (v == s_cur_res[WID][s_v_order_[l]])
          if (v == s_cur_res[WID][l])
          {
            dup = true; //重复
            break;
          }
        }
      __syncwarp();
      if (v != UINT32_MAX &&
          !dup &&
          // (d_encodings_[v * num_blocks + s_pos_u[s_u[WID]] / BLK_SIZE] & (1 << (s_pos_u[s_u[WID]] % BLK_SIZE)))
          d_encodings_[s_u[WID] * C_COL_LEN + v / BLK_SIZE] & (1 << (v % BLK_SIZE)))
      {
        int mask = __activemask();// 获取当前活跃线程的掩码
        int size = __popc(mask);// 当前活跃线程数
        int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;// 计算当前线程在活跃线程中的排名
        if (rank == 0)
        { //原子分配当前warp的结果起始位置
          warp_pos[WID] = atomicAdd(num_res_new, size);
          if (warp_pos[WID] + size >= MAX_RES)
          {
            *exceed = 1;
          }
        }
        __syncwarp(mask);
        if (*exceed == 1)
          return;
        int my_pos = warp_pos[WID] + rank;
        // 需要前面的多一个所以要加2
        d_res_table[my_pos * (level + 2)] = s_order_id[WID];
        for (int i = 0; i < level; ++i)
          d_res_table[my_pos * (level + 2) + i + 1] = s_cur_res[WID][i];
        d_res_table[my_pos * (level + 2) + level + 1] = v;
        // for (int i = 0; i < C_NUM_VQ; ++i)
        // d_res_table[my_pos * C_NUM_VQ + i] = d_res_table_old[s_row[WID] * C_NUM_VQ + i];
        // d_res_table[my_pos * C_NUM_VQ + s_u[WID]] = v;
      }
      __syncwarp();
      off += warpSize;
    }
    if (LID == 0)
      block_iter_cnt[WID]++;
    __syncwarp();
  }
}

void parallelMatch(
    cpuGraph *hq, cpuGraph *hg,
    gpuGraph *dq, gpuGraph *dg,
    OrderCPU *h_order_obj,

    uint32_t *d_compact_encodings_,
    encodingMeta *enc_meta,
    uint32_t *d_u_candidate_vs_, numtype *d_num_u_candidate_vs_,
    numtype *h_num_u_candidate_vs_,

    ResTable *res_table,std::vector<LocalView> local_views)
{
  // gpuGraph *real_dg;
  // cuchk(hipMalloc((void **)&real_dq, sizeof(gpuGraph)));
  // cuchk(hipMalloc((void **)&real_dg, sizeof(gpuGraph)));

  // cuchk(hipMemcpy(real_dq, dq, sizeof(gpuGraph), hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(real_dg, dg, sizeof(gpuGraph), hipMemcpyHostToDevice));

  // std::cout << "free memory: " << getFreeGlobalMemory(GPU_NUM) << std::endl;
  // std::cout << "MAX_L_FREQ: " << MAX_L_FREQ << std::endl;
  // std::cout << "NUM_VQ: " << NUM_VQ << std::endl;

  NUM_CAN_UB = 0;
  for (int i = 0; i < NUM_VQ; ++i)
    NUM_CAN_UB = std::max(NUM_CAN_UB, h_num_u_candidate_vs_[i]);
  NUM_CAN_UB = std::min(NUM_CAN_UB, MAX_DATA_DEGREE);
  hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_CAN_UB), &NUM_CAN_UB, sizeof(uint32_t));
  // std::cout << "NUM_CAN_UB: " << NUM_CAN_UB << std::endl;

  /*--- tdfs ---*/
  CallStack *callstack_gpu;
  std::vector<CallStack> stk(NWARPS_TOTAL);

  vtype *candidate_space;
  cuchk(hipMalloc((void **)&candidate_space, sizeof(vtype) * NUM_VQ * NUM_CAN_UB * NWARPS_TOTAL));

  for (int i = 0; i < NWARPS_TOTAL; i++)
  {
    auto &s = stk[i];
    s.candidates_ = candidate_space + i * NUM_CAN_UB * NUM_VQ;
    memset(s.iter, 0, sizeof(vtype) * MAX_VQ);
    memset(s.num_candidates_, 0, sizeof(numtype) * MAX_VQ);
    // memset(s.map_res_, 0, sizeof(s.map_res_));
  }
  cuchk(hipMalloc(&callstack_gpu, NWARPS_TOTAL * sizeof(CallStack)));
  cuchk(hipMemcpy(callstack_gpu, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice));

  int *cur;
  cuchk(hipMalloc((void **)&cur, sizeof(int)));
  cuchk(hipMemset(cur, 0, sizeof(int)));

  unsigned long long *d_res;
  cuchk(hipMalloc((void **)&d_res, sizeof(unsigned long long) * NWARPS_TOTAL));
  cuchk(hipMemset(d_res, 0, sizeof(unsigned long long) * NWARPS_TOTAL));

  numtype &num_orders = h_order_obj->num_orders;
  OrderGPU order_gpu_temp(num_orders);
  cuchk(hipMemcpy(order_gpu_temp.num_orders, &num_orders, sizeof(numtype), hipMemcpyHostToDevice));
  // order_gpu_temp.num_orders = num_orders;
  cuchk(hipMemcpy(order_gpu_temp.roots_, h_order_obj->roots.data(), sizeof(vtype) * num_orders, hipMemcpyHostToDevice));
  for (int i = 0; i < num_orders; ++i)
  {
    cuchk(hipMemcpy(order_gpu_temp.v_orders_ + i * NUM_VQ, h_order_obj->v_orders[i].data(), sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
    cuchk(hipMemcpy(order_gpu_temp.u2ls_ + i * NUM_VQ, h_order_obj->u2ls[i].data(), sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));
    cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_ + i * NUM_VQ, h_order_obj->num_backward_neighbors[i].data(), sizeof(numtype) * NUM_VQ, hipMemcpyHostToDevice));
  }
  // cuchk(hipMemcpy(order_gpu_temp.v_orders_, h_order_obj->v_orders.data(), sizeof(vtype) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.u2ls_, h_order_obj->u2ls.data(), sizeof(int) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_, h_order_obj->num_backward_neighbors.data(), sizeof(numtype) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  offtype off = 0;
  for (int i = 0; i < num_orders; ++i)
  {
    for (int j = 0; j < NUM_VQ; ++j)
    {
      cuchk(hipMemcpy(order_gpu_temp.backward_neighbors_ + off, h_order_obj->backward_neighbors[i][j].data(), sizeof(vtype) * h_order_obj->num_backward_neighbors[i][j], hipMemcpyHostToDevice));
      off += NUM_VQ;
    }
  }

  // cuchk(hipMemcpy(order_gpu_temp.root_u, &h_order_obj->root_u, sizeof(vtype), hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.v_order_, h_order_obj->v_order_, sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.u2l_, h_order_obj->u2l_, sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.e_order_, h_order_obj->e_order_, sizeof(etype) * NUM_EQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.shared_neighbors_with_, h_order_obj->shared_neighbors_with_, sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_, h_order_obj->num_backward_neighbors_, sizeof(numtype) * NUM_VQ, hipMemcpyHostToDevice));
  // offtype off = 0;
  // for (int i = 0; i < NUM_VQ; ++i)
  // {
  // cuchk(hipMemcpy(order_gpu_temp.backward_neighbors_ + off, h_order_obj->backward_neighbors_[i], sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // off += NUM_VQ;
  // }

  /*--- order, encoding ---*/
  OrderGPU *real_order_gpu;
  cuchk(hipMalloc((void **)&real_order_gpu, sizeof(OrderGPU)));
  cuchk(hipMemcpy(real_order_gpu, &order_gpu_temp, sizeof(OrderGPU), hipMemcpyHostToDevice));

  numtype &num_blocks = enc_meta->num_blocks;

  // vtype u = h_order_obj->v_order_[0];

  vtype *d_res_table_old_;
  // unsigned long long h_num_res_old = h_num_u_candidate_vs_[u];
  unsigned long long h_num_res_old;
  cuchk(hipMalloc((void **)&d_res_table_old_, sizeof(vtype) * NUM_VQ * MAX_RES));

  vtype *d_res_table_;
  unsigned long long h_num_res_new = 0;
  cuchk(hipMalloc((void **)&d_res_table_, sizeof(vtype) * NUM_VQ * MAX_RES));

  int start_level = 0; // at `start_level`, tasks are done, just fetch from table. real match begins from `start_level + 1`.

  vtype *d_intersect_temp_storage;
  cuchk(hipMalloc((void **)&d_intersect_temp_storage, sizeof(vtype) * NUM_CAN_UB * NWARPS_TOTAL));
  numtype *d_num_intersect_temp_storage;
  cuchk(hipMalloc((void **)&d_num_intersect_temp_storage, sizeof(numtype) * NWARPS_TOTAL));
  cuchk(hipMemset(d_num_intersect_temp_storage, 0, sizeof(numtype) * NWARPS_TOTAL));

  warmup<<<GRID_DIM, BLOCK_DIM>>>();
  cuchk(hipDeviceSynchronize());

  TIME_INIT();
  TIME_START();

  // struct timespec time_st;
  // struct timespec time_ed;

  micro_init();
  micro_start();

  // clock_gettime(CLOCK_REALTIME, &time_st);
  // dim3 fj_block = BLOCK_DIM;
  // dim3 fj_grid = (h_num_res_old - 1) / fj_block.x + 1;
  // firstJoinKernel<<<fj_grid, fj_block>>>(u, d_u_candidate_vs_, h_num_u_candidate_vs_[u], d_res_table_old_);
  // cuchk(hipDeviceSynchronize());
  // std::cout << "first join done" << std::endl;

  // hipFree(d_u_candidate_vs_);
  // hipFree(d_num_u_candidate_vs_);

  int level = 2;
  cuchk(hipMemcpy(d_res_table_old_, res_table->res_table, sizeof(vtype) * res_table->size, hipMemcpyHostToDevice));
  h_num_res_old = res_table->size / (level + 1); // 4 有多少组不同的匹配

#ifndef NDEBUG
  std::cout << "res_table: " << std::endl;
  for (int i = 0; i < res_table->size; ++i)
  {
    std::cout << res_table->res_table[i] << " ";
  }
#endif

 

  offtype v_off = 2;

  int *d_enc_pos_u_;
  // cuchk(hipMalloc((void **)&d_enc_pos_u_, sizeof(int) * NUM_VQ));
  // cuchk(hipMemcpy(d_enc_pos_u_, enc_meta->enc_pos_of_u_, sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));

  numtype *d_num_res_new;
  cuchk(hipMalloc((void **)&d_num_res_new, sizeof(numtype)));
  cuchk(hipMemset(d_num_res_new, 0, sizeof(numtype)));

  int *exceed;
  cuchk(hipMalloc((void **)&exceed, sizeof(int)));
  cuchk(hipMemset(exceed, 0, sizeof(int)));

  int h_exceed; //BFS
  while (v_off < NUM_VQ)
  {
    // printf("-------");
    //printf("当前BFS层数为%d\n",v_off);
    vertexJoinBFS<<<GRID_DIM, BLOCK_DIM>>>(
        dg->offsets_, dg->neighbors_, dg->degree_,
        real_order_gpu,
        v_off,
        d_intersect_temp_storage, d_num_intersect_temp_storage,

        d_compact_encodings_, num_blocks, d_enc_pos_u_,
        d_res_table_old_, h_num_res_old,
        d_res_table_, d_num_res_new,
        exceed);
    cuchk(hipDeviceSynchronize());
    //printf("当前BFS层数%djieshu\n",v_off);
    cuchk(hipMemcpy(&h_exceed, exceed, sizeof(int), hipMemcpyDeviceToHost));
    if (h_exceed == 1)
    {
      std::cout << "exceed" << std::endl;
      break;
    }

    // auto num_res_backup = h_num_res_old;
    cuchk(hipMemcpy(&h_num_res_old, d_num_res_new, sizeof(numtype), hipMemcpyDeviceToHost));
    std::swap(d_res_table_old_, d_res_table_);
    cuchk(hipMemset(d_num_res_new, 0, sizeof(numtype)));

    v_off++;
  }

  micro_end();
  micro_print_local("bfs");
  total_match_time_us += diff_micro.tv_nsec / 1000.0;

  TIME_END();
  total_match_time_ms += kernel_time;

  // clock_gettime(CLOCK_REALTIME, &time_ed);

  // std::cout << "bfs: " << time_ed.tv_nsec - time_st.tv_nsec << "(ns)" << std::endl;
  // TIME_END();
  // PRINT_LOCAL_TIME("BFS");

  cuchk(hipFree(d_res_table_));
  if (v_off == NUM_VQ)
  {
    std::cout << "res: " << h_num_res_old << std::endl;
    std::cout << std::endl;
    return;
  }
  // std::cout << "u: " << h_order_obj->v_order_[start_level] << " num_rows: " << h_num_res_old << std::endl;

#ifndef NDEBUG
  std::cout << "prepare done, entering match kernel" << std::endl;
#endif

  start_level = v_off;
  if (start_level >= NUM_VQ - 1)
  {
    // std::cout << "res: " << h_num_res_old << std::endl;
    return;
  }

  // STOP_LEVEL = std::max((uint32_t)start_level + 1, NUM_VQ / 2);
  // start_level--;
  STOP_LEVEL = start_level + 2;
  cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_STOP_LEVEL), &STOP_LEVEL, sizeof(uint32_t)));

  int *gpu_timeout_queue_space;
  cuchk(hipMalloc(&gpu_timeout_queue_space, sizeof(int) * TIMEOUT_QUEUE_CAP * STOP_LEVEL));
  // vtype *gpu_timeout_candidate_queue_space;
  // hipMalloc(&gpu_timeout_candidate_queue_space, sizeof(vtype) * TIMEOUT_QUEUE_CAP * WARP_SIZE);
  // hipMemset(gpu_timeout_candidate_queue_space, UINT32_MAX, sizeof(vtype) * TIMEOUT_QUEUE_CAP * WARP_SIZE);
  Queue *gpu_timeout_queue;
  hipMallocManaged(&gpu_timeout_queue, sizeof(Queue));
  gpu_timeout_queue->queue_ = gpu_timeout_queue_space;
  // gpu_timeout_queue->candidate_queue = gpu_timeout_candidate_queue_space;
  gpu_timeout_queue->size_ = TIMEOUT_QUEUE_CAP * (STOP_LEVEL);
  gpu_timeout_queue->resetQueue();
  hipMemPrefetchAsync(gpu_timeout_queue, sizeof(Queue), GPU_NUM);
  hipDeviceSynchronize();

  // size_t shared_used = 0;
  // shared_used += sizeof(OrderGPU);
  // shared_used += sizeof(gpuGraph);
  // shared_used += sizeof(CallStack) * WARP_PER_BLOCK;
  // shared_used += sizeof(unsigned long long) * WARP_PER_BLOCK;
  // shared_used += sizeof(int) * MAX_VQ;
  // shared_used += sizeof(vtype) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(vtype) * MAX_VQ;
  // shared_used += sizeof(Arg_t) * WARP_PER_BLOCK;
  // shared_used += sizeof(bool) * WARP_PER_BLOCK;
  // shared_used += sizeof(vtype) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(int) * WARP_PER_BLOCK;
  // shared_used += sizeof(int) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(StealingArgs);
  // std::cout << shared_used * 1.0 / 1024 << "KB" << std::endl;
  // std::cout << "totally: " << shared_used * 1.0 / 1024 * GRID_DIM << "KB" << std::endl;

  // TIME_INIT();
  TIME_START();
  micro_start();

  // clock_gettime(CLOCK_REALTIME, &time_st);

  parallel_match_kernel<<<GRID_DIM, BLOCK_DIM>>>(
      dg->degree_, dg->offsets_, dg->neighbors_,
      callstack_gpu, cur, d_res,
      gpu_timeout_queue, real_order_gpu, h_order_obj->num_orders,

      start_level,

      d_compact_encodings_,
      enc_meta->num_blocks,
      d_u_candidate_vs_, d_num_u_candidate_vs_,

      d_res_table_old_, h_num_res_old);
  cuchk(hipDeviceSynchronize());

  micro_end();
  micro_print_local("dfs");
  total_match_time_us += diff_micro.tv_nsec / 1000.0;
  // clock_gettime(CLOCK_REALTIME, &time_ed);

  // std::cout << "dfs: " << time_ed.tv_nsec - time_st.tv_nsec << "(ns)" << std::endl;

  TIME_END();
  PRINT_LOCAL_TIME("DFS_JOIN");
  total_match_time_ms += kernel_time;
  // PRINT_TOTAL_TIME("HYBRID_JOIN");
  // std::cout << "Parallel_match_kernel done" << std::endl;

  unsigned long long *h_res = new unsigned long long[NWARPS_TOTAL];
  cuchk(hipMemcpy(h_res, d_res, sizeof(unsigned long long) * NWARPS_TOTAL, hipMemcpyDeviceToHost));
  unsigned long long res = 0;
  for (int i = 0; i < NWARPS_TOTAL; ++i)
    res += h_res[i];
  std::cout << "res: " << res << std::endl;
  std::cout << std::endl;
}